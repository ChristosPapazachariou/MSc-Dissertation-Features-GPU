#include "hip/hip_runtime.h"
/***********************************************************************************************************************
This file is part of "SIFT-GPU-Thesis" project.
 * Copyright (C) 2015 {Christos Papazachariou} <{christospapazachariou@gmail.com}>
 * University of Piraeus- MSc program: Advanced Information Systems ("Embedded System Technologies" branch).
 * Partial or otherwise use of this project and/or source code is permitted only for educational and academic purposes.
 * Partial or otherwise use of this project and/or source code for commercial or militay applications is prohibited.
***********************************************************************************************************************/

#include "sift.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

extern "C" {
#include "GPUDummy.h"
}


extern "C"{
	__global__ void GPUDummyKernel(unsigned char* GPUoutput, int inH, int inW){

		int y=blockIdx.x;
		int x=threadIdx.x;

		GPUoutput[x*inW+y]= (unsigned char)((int)((x*y)/256));
					
	}
}

extern "C"{
	void dummyWrapper(unsigned char* output,int inH, int inW){
		unsigned char* GPUtemp;

		if(hipMalloc(&GPUtemp,sizeof(unsigned char)*inW*inH)!=hipSuccess){
			printf ("hipMalloc Error!\n");	
			getchar();
			hipFree(GPUtemp);
		}else{
			printf ("hipMalloc Success!\n");	
		}

		if(hipMemcpy(GPUtemp, output, sizeof(unsigned char)*inW*inH, hipMemcpyHostToDevice)!=hipSuccess){ 
			printf ("hipMemcpyHostToDevice Error!\n");	
			getchar();
			hipFree(GPUtemp);
		}else{
			printf ("hipMemcpyHostToDevice Success!\n");	
		}

		printf("dummy CUDA kernel call.. \n");

		GPUDummyKernel<<<inH,					//blocks=height of image
						inW>>>					//threads per block = width of image(max 1024)
						(GPUtemp,				//pointer to destination
						inW, inH);				//destination dimensions

		if(hipMemcpy(output, GPUtemp, sizeof(unsigned char)*inW*inH, hipMemcpyDeviceToHost)!=hipSuccess){
			printf ("hipMemcpyDeviceToHost Error!\n");	
			getchar();
			hipFree(GPUtemp);
		}else{
			printf ("hipMemcpyDeviceToHost Success!\n");	
		}

		hipFree(GPUtemp);
	}
}
